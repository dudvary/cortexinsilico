#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <map>
#include <set>
#include <vector>
#include <string>
#include <iostream>
#include <windows.h>
#include <fstream>
#include <ctime>
#include <random>
#include <omp.h>
#include <fstream>

/*
#####################################################################################
READ FEATURES
#####################################################################################
*/

std::string getBaseDir() {
	char buffer[MAX_PATH];
	GetModuleFileName(NULL, buffer, MAX_PATH);
	std::string::size_type pos = std::string(buffer).find_last_of("\\/");
	return std::string(buffer).substr(0, pos);
}

std::string getDirPath(const std::string baseDir, const std::string dirName) {
	std::string path = baseDir;
	path.append("\\");
	path.append(dirName);
	return path;
}

std::vector<std::string> getFiles(const std::string& dir)
{
	std::vector<std::string> v;
	std::string pattern(dir);
	pattern.append("\\*.dat");
	WIN32_FIND_DATA data;
	HANDLE hFind;
	if ((hFind = FindFirstFile(pattern.c_str(), &data)) != INVALID_HANDLE_VALUE) {
		do {
			std::string path = dir;
			path.append("\\");
			path.append(data.cFileName);
			v.push_back(path);
		} while (FindNextFile(hFind, &data) != 0);
		FindClose(hFind);
	}
	return v;
}

void readMapFloat(const std::string filePath, std::map<int, float>& voxel_value, float coefficient) {
	std::ifstream infile(filePath);
	int voxelId;
	float value;
	while (infile >> voxelId >> value)
	{
		voxel_value[voxelId] = coefficient * value;
	}
}

int getIndexFromPath(const std::string path) {
	std::size_t pos1 = path.find_last_of("\\");
	std::string fileName = path.substr(pos1 + 1);
	return std::stoi(fileName);
}

void readFields(const std::vector<std::string>& filePaths, std::map<int, std::map<int, float> >& neuron_voxel_value, float coefficient) {
	for (auto it = filePaths.begin(); it != filePaths.end(); ++it) {
		int neuronId = getIndexFromPath(*it);
		std::map<int, float> voxel_value;
		readMapFloat(*it, voxel_value, coefficient);
		neuron_voxel_value[neuronId] = voxel_value;
	}
}

/*
#####################################################################################
PREPARE DATA
#####################################################################################
*/

void flattenFeatures(std::map<int, std::map<int, float> >& neuron_voxel_pre,
	std::map<int, std::map<int, float> >& neuron_voxel_postExc,
	std::map<int, float>& voxel_postAllExc,
	float* pre,
	float* post,
	float* postAll) {
	int i = 0;
	int nVoxel = voxel_postAllExc.size();
	for (auto it = voxel_postAllExc.begin(); it != voxel_postAllExc.end(); ++it) {
		int voxelId = it->first;
		int j = 0;
		for (auto it2 = neuron_voxel_pre.begin(); it2 != neuron_voxel_pre.end(); ++it2) {
			auto x = it2->second.find(voxelId);
			if (x != it2->second.end()) {
				pre[j * nVoxel + i] = x->second;
			}
			else {
				pre[j * nVoxel + i] = 0;
			}
			j++;
		}
		j = 0;
		for (auto it2 = neuron_voxel_postExc.begin(); it2 != neuron_voxel_postExc.end(); ++it2) {
			auto x = it2->second.find(voxelId);
			if (x != it2->second.end()) {
				post[j * nVoxel + i] = x->second;
			}
			else {
				post[j * nVoxel + i] = 0;
			}
			j++;
		}
		postAll[i] = it->second;
		i++;
	}
}

/*
#####################################################################################
GPU COMPUTATION
#####################################################################################
*/

__global__ void calcKernel(float *contacts,
	float *pre,
	float *post,
	float* postAll,
	float b0,
	unsigned int nVoxel,
	unsigned int nPre,
	unsigned int nPost
)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < nPre && j < nPost) {
		for (unsigned int k = 0; k < nVoxel; k++)
		{
			float preVal = pre[i * nVoxel + k];
			float postVal = post[j * nVoxel + k];
			if (preVal != 0 && postVal != 0 && contacts[i * nPost + j] < 1000) {
				float arg = b0 + preVal + postVal + postAll[k];
				int synapses = 0;
				if (arg >= -7 && arg <= 7)
				{
					float mu = exp(arg);
					contacts[i * nPost + j] += mu;
				}
				else if (arg > 7)
				{
					contacts[i * nPost + j] = 1000;
				}
			}
		}
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t calcWithCuda(float *pre,
	float *post,
	float *postAll,
	float b0,
	unsigned int nPre,
	unsigned int nPost,
	unsigned int nVoxel,
	float* contacts,
	std::clock_t start,
	bool verbose)
{
	float *dev_pre = 0;
	float *dev_post = 0;
	float *dev_postAll = 0;
	float *dev_contacts = 0;
	hipError_t cudaStatus;

	double copyToDeviceStartTime = std::clock();

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pre, nPre * nVoxel * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_post, nPost * nVoxel * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_postAll, nVoxel * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_contacts, nPre * nPost * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_pre, pre, nPre * nVoxel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_post, post, nPost * nVoxel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_postAll, postAll, nVoxel * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_contacts, contacts, nPre * nPost * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	double duration;
	if (verbose) {
		duration = (std::clock() - copyToDeviceStartTime) / (double)CLOCKS_PER_SEC;
		std::cout << "[*] Copy to device " << duration << std::endl;
	}
	double computeKernelStartTime = std::clock();

	// Launch a kernel on the GPU
	dim3 threads(16, 16);
	dim3 blocks(nPre / threads.x + 1, nPost / threads.y + 1);
	calcKernel << <blocks, threads >> > (dev_contacts,
		dev_pre,
		dev_post,
		dev_postAll,
		b0,
		nVoxel,
		nPre,
		nPost
		);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calcKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	if (verbose) {
		duration = (std::clock() - computeKernelStartTime) / (double)CLOCKS_PER_SEC;
		std::cout << "[*] Compute GPU " << duration << std::endl;
	}
	double copyToHostTime = std::clock();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(contacts, dev_contacts, nPre * nPost * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	if (verbose) {
		duration = (std::clock() - copyToHostTime) / (double)CLOCKS_PER_SEC;
		std::cout << "[*] Copy to host " << duration << std::endl;
	}
Error:
	hipFree(dev_pre);
	hipFree(dev_post);
	hipFree(dev_postAll);
	hipFree(dev_contacts);

	return cudaStatus;
}

/*
#####################################################################################
MAIN
#####################################################################################
*/

int main(int argc, char *argv[]) {

	if (argc != 6) {
		std::cout << "Usage:" << std::endl;
		std::cout << "Simulator.exe CPU|GPU theta1 theta2 theta3 theta4" << std::endl;
		return -1;
	}

	std::string mode = argv[1];
	bool gpu = mode.compare("GPU") == 0;

	float b0, b1, b2, b3;
	b0 = std::stof(argv[2]);
	b1 = std::stof(argv[3]);
	b2 = std::stof(argv[4]);
	b3 = std::stof(argv[5]);

	bool verbose = false;

	std::cout << "[*] Start simulation " << b0 << " " << b1 << " " << b2 << " " << b3 << std::endl;
	std::clock_t start;
	double duration;
	start = std::clock();

	std::map<int, std::map<int, float> > neuron_voxel_pre;
	std::map<int, std::map<int, float> > neuron_voxel_postExc;
	std::map<int, float> voxel_postAllExc;

	std::string baseDir = getBaseDir();

	double readFeatureTime = std::clock();

	std::vector<std::string> preFiles = getFiles(getDirPath(baseDir, "features_pre"));
	readFields(preFiles, neuron_voxel_pre, b1);
	readFields(getFiles(getDirPath(baseDir, "features_postExc")), neuron_voxel_postExc, b2);
	readMapFloat(getDirPath(baseDir, "features_postAll").append("\\voxel_postAllExc.dat"), voxel_postAllExc, b3);

	if (verbose) {
		duration = (std::clock() - readFeatureTime) / (double)CLOCKS_PER_SEC;
		std::cout << "[*] Read features " << duration << std::endl;
	}
	std::size_t nPre = neuron_voxel_pre.size();
	std::size_t nPost = neuron_voxel_postExc.size();
	std::size_t nVoxel = voxel_postAllExc.size();

	if (verbose) {
		std::cout << "[*] Presynaptic: " << nPre << " Postsynaptic: " << nPost << " Voxels: " << nVoxel << std::endl;
	}
	std::random_device rd;
	std::mt19937 randomGenerator(rd());
	std::uniform_real_distribution<float> dis(0.0, 1.0);

	float* connections = (float*)malloc(nPre * nPost * sizeof(float));

	for (int i = 0; i < nPre; i++) {
		for (int j = 0; j < nPost; j++) {
			connections[i * nPost + j] = 0;
		}
	}

	if (!gpu) {

		std::vector<int> preIndices;
		std::vector<int> postIndices;

		for (auto it = neuron_voxel_pre.begin(); it != neuron_voxel_pre.end(); ++it)
		{
			preIndices.push_back(it->first);
		}

		for (auto it = neuron_voxel_postExc.begin(); it != neuron_voxel_postExc.end(); ++it)
		{
			postIndices.push_back(it->first);
		}

		double computeCPUStartTime = std::clock();

#pragma omp parallel for schedule(dynamic)
		for (unsigned int i = 0; i < preIndices.size(); i++)
		{
			int preId = preIndices[i];
			for (unsigned int j = 0; j < postIndices.size(); j++)
			{
				int postId = postIndices[j];
				//qDebug() << i << j << preId << postId;
				if (preId != postId)
				{
					for (auto pre = neuron_voxel_pre[preId].begin(); pre != neuron_voxel_pre[preId].end(); ++pre)
					{
						if (neuron_voxel_postExc[postId].find(pre->first) != neuron_voxel_postExc[postId].end())
						{
							float preVal = pre->second;
							float postVal = neuron_voxel_postExc[postId][pre->first];
							float postAllVal = voxel_postAllExc[pre->first];
							float arg = b0 + preVal + postVal + postAllVal;
							//int synapses = 0;
							if (arg >= -7 && arg <= 7)
							{
								float mu = exp(arg);
								connections[i * nPost + j] += mu;
							}
							else if (arg > 7)
							{
								connections[i * nPost + j] = 1000;
								break;
							}
						}
					}
				}
			}
		}

		if (verbose) {
			duration = (std::clock() - computeCPUStartTime) / (double)CLOCKS_PER_SEC;
			std::cout << "[*] Compute CPU " << duration << std::endl;
		}
	}
	else
	{
		float* pre = (float*)malloc(nPre * nVoxel * sizeof(float));
		float* post = (float*)malloc(nPost * nVoxel * sizeof(float));
		float* postAll = (float*)malloc(nVoxel * sizeof(float));

		double flattenFeaturesTime = std::clock();

		flattenFeatures(neuron_voxel_pre, neuron_voxel_postExc, voxel_postAllExc, pre, post, postAll);

		if (verbose) {
			duration = (std::clock() - flattenFeaturesTime) / (double)CLOCKS_PER_SEC;
			std::cout << "[*] Flatten features " << duration << std::endl;
		}
		hipError_t cudaStatus = calcWithCuda(pre, post, postAll, b0, nPre, nPost, nVoxel, connections, start, verbose);

		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calcWithCuda failed!");
			return 1;
		}

		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		free(pre);
		free(post);
		free(postAll);
	}

	double computeProbabilityStartTime = std::clock();

	float connectionProbability = 0;
	for (unsigned int i = 0; i < nPre; i++) {
		int realizedConnections = 0;
		for (unsigned int j = 0; j < nPost; j++) {
			float mu = connections[i * nPost + j];
			if (mu <= 1000) {
				float prob = 1 - exp(-1 * mu);
				float rand = dis(randomGenerator);
				realizedConnections += rand <= prob ? 1 : 0;
			}
			else {
				realizedConnections++;
			}
		}
		connectionProbability += (float)realizedConnections / (float)nPost;
	}
	connectionProbability /= (float)nPre;
	free(connections);

	duration = (std::clock() - computeProbabilityStartTime) / (double)CLOCKS_PER_SEC;
	//std::cout << "[*] Compute connection probability " << duration << std::endl;

	std::ofstream outfile;
	outfile.open("output.json");
	outfile << "{\"CONNECTION_PROBABILITY\":" << connectionProbability << "}";
	outfile.close();

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "[*] Finish simulation " << duration << "s" << std::endl;
	std::cout << "[*] Connection prob. " << connectionProbability << std::endl;


	return 0;
}




