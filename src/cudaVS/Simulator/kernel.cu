#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <map>
#include <set>
#include <vector>
#include <string>
#include <iostream>
#include <windows.h>
#include <fstream>
#include <ctime>
#include <random>

/*
#####################################################################################
READ FEATURES
#####################################################################################
*/

std::string getBaseDir() {
	char buffer[MAX_PATH];
	GetModuleFileName(NULL, buffer, MAX_PATH);
	std::string::size_type pos = std::string(buffer).find_last_of("\\/");
	return std::string(buffer).substr(0, pos);
}

std::string getDirPath(const std::string baseDir, const std::string dirName) {
	std::string path = baseDir;
	path.append("\\");
	path.append(dirName);
	return path;
}

std::vector<std::string> getFiles(const std::string& dir)
{
	std::vector<std::string> v;
	std::string pattern(dir);
	pattern.append("\\*.dat");
	WIN32_FIND_DATA data;
	HANDLE hFind;
	if ((hFind = FindFirstFile(pattern.c_str(), &data)) != INVALID_HANDLE_VALUE) {
		do {
			std::string path = dir;
			path.append("\\");
			path.append(data.cFileName);
			v.push_back(path);
		} while (FindNextFile(hFind, &data) != 0);
		FindClose(hFind);
	}
	return v;
}

void readMapFloat(const std::string filePath, std::map<int, float>& voxel_value, float coefficient) {
	std::ifstream infile(filePath);
	int voxelId;
	float value;
	while (infile >> voxelId >> value)
	{
		voxel_value[voxelId] = coefficient * value;
	}
}

int getIndexFromPath(const std::string path) {
	std::size_t pos1 = path.find_last_of("\\");
	std::string fileName = path.substr(pos1 + 1);
	return std::stoi(fileName);
}

void readFields(const std::vector<std::string>& filePaths, std::map<int, std::map<int, float> >& neuron_voxel_value, float coefficient) {
	for (auto it = filePaths.begin(); it != filePaths.end(); ++it) {
		int neuronId = getIndexFromPath(*it);
		std::map<int, float> voxel_value;
		readMapFloat(*it, voxel_value, coefficient);
		neuron_voxel_value[neuronId] = voxel_value;
	}
}

/*
#####################################################################################
PREPARE DATA
#####################################################################################
*/

void flattenFeatures(std::map<int, std::map<int, float> >& neuron_voxel_pre,
	std::map<int, std::map<int, float> >& neuron_voxel_postExc,
	std::map<int, float>& voxel_postAllExc,
	float* pre,
	float* post,
	float* postAll) {
	int i = 0;
	int nVoxel = voxel_postAllExc.size();
	for (auto it = voxel_postAllExc.begin(); it != voxel_postAllExc.end(); ++it) {
		int voxelId = it->first;
		int j = 0;
		for (auto it2 = neuron_voxel_pre.begin(); it2 != neuron_voxel_pre.end(); ++it2) {
			auto x = it2->second.find(voxelId);
			if (x != it2->second.end()) {
				pre[j * nVoxel + i] = x->second;
			}
			else {
				pre[j * nVoxel + i] = 0;
			}
			j++;
		}
		j = 0;
		for (auto it2 = neuron_voxel_postExc.begin(); it2 != neuron_voxel_postExc.end(); ++it2) {
			auto x = it2->second.find(voxelId);
			if (x != it2->second.end()) {
				post[j * nVoxel + i] = x->second;
			}
			else {
				post[j * nVoxel + i] = 0;
			}
			j++;
		}
		postAll[i] = it->second;
		i++;
	}
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	//addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

/*
#####################################################################################
MAIN
#####################################################################################
*/

int main(int argc, char *argv[]) {

	/*
	if (argc != 5) {
		std::cout << "Usage:" << std::endl;
		std::cout << "Simulator.exe theta1 theta2 theta3 theta4" << std::endl;
		return -1;
	}*/

	float b0, b1, b2, b3;
	b0 = 0;//std::stof(argv[1]);
	b1 = 1;// std::stof(argv[2]);
	b2 = 1;// std::stof(argv[3]);
	b3 = -1;// std::stof(argv[4]);

	std::cout << "Start simulation" << " " << b0 << " " << b1 << " " << b2 << " " << b3 << std::endl;
	std::clock_t start;
	double duration;
	start = std::clock();

	std::map<int, std::map<int, float> > neuron_voxel_pre;
	std::map<int, std::map<int, float> > neuron_voxel_postExc;
	std::map<int, float> voxel_postAllExc;

	std::string baseDir = getBaseDir();

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Start list files pre" << " " << duration << std::endl;

	std::vector<std::string> preFiles = getFiles(getDirPath(baseDir, "features_pre"));

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Listed files pre" << " " << duration << std::endl;

	readFields(preFiles, neuron_voxel_pre, b1);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Read features pre" << " " << duration << std::endl;
	
	readFields(getFiles(getDirPath(baseDir, "features_postExc")), neuron_voxel_postExc, b2);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Read features post" << " " << duration << std::endl;

	readMapFloat(getDirPath(baseDir, "features_postAll").append("\\voxel_postAllExc.dat"), voxel_postAllExc, b3);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Read data" << " " << duration << std::endl;

	std::size_t nPre = neuron_voxel_pre.size();
	std::size_t nPost = neuron_voxel_postExc.size();
	std::size_t nVoxel = voxel_postAllExc.size();

	float* pre = (float*)malloc(nPre * nVoxel * sizeof(float));
	float* post = (float*)malloc(nPost * nVoxel * sizeof(float));
	float* postAll = (float*)malloc(nVoxel * sizeof(float));

	flattenFeatures(neuron_voxel_pre, neuron_voxel_postExc, voxel_postAllExc, pre, post, postAll);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Flattened data" << " " << duration << std::endl;

	std::random_device rd;
	std::mt19937 randomGenerator(rd());

	std::vector<int> empty(nPost, 0);
	std::vector<std::vector<int> > contacts(nPre, empty);
	std::uniform_real_distribution<float> dis(0.0, 1.0);

	#pragma omp parallel for schedule(dynamic)
	for (unsigned int i = 0; i < nPre; i++)
	{
		for (unsigned int j = 0; j < nPost; j++)
		{
			for (unsigned int k = 0; k < nVoxel; k++)
			{
				float preVal = pre[i * nVoxel + k];
				float postVal = post[j * nVoxel + k];
				float postAllVal = postAll[k];
				if (preVal != 0 && postVal != 0) {
					float arg = b0 + preVal + postVal + postAllVal;
					int synapses = 0;
					if (arg >= -7 && arg <= 7)
					{
						float mu = exp(arg);
						float prob = 1 - exp(-1 * mu);
						float rand = dis(randomGenerator);
						synapses = rand <= prob ? 1 : 0;
					}
					else if (arg > 7)
					{
						synapses = 1;
					}
					if (synapses > 0)
					{
						contacts[i][j] = synapses;
						break;
					}
				}
			}
		}
	}

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Finished synapse distribution" << " " << duration << std::endl;

	float connectionProbability = 0;

	for (unsigned int i = 0; i < nPre; i++) {
		int realizedConnections = 0;
		for (unsigned int j = 0; j < nPost; j++) {
			realizedConnections += contacts[i][j] > 0 ? 1 : 0;
		}
		connectionProbability += (float)realizedConnections / (float)nPost;
	}
	connectionProbability /= (float)nPre;

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Computed connection probability" << " " << duration << std::endl;

	/*
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	*/

	/*
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	*/

	std::cout << "Finish simulation" << " " << b0 << " " << b1 << " " << b2 << " " << b3 << " prob. " << connectionProbability << std::endl;

	free(pre);
	free(post);
	free(postAll);

	return 0;
}




