#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <QChar>
#include <QDebug>
#include <QFile>
#include <QIODevice>
#include <QList>
#include <QString>
#include <QStringList>
#include <QTextStream>
#include <QDirIterator>

#include <stdio.h>
#include <map>
#include <set>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void
readMapFloat(std::map<int, float>& mapping, QString folder, QString fileName, float coefficient)
{
	if (folder != "")
	{
		fileName = QDir(folder).filePath(fileName);
	}
	QFile file(fileName);
	if (!file.open(QIODevice::ReadOnly | QIODevice::Text))
	{
		const QString msg =
			QString("Error reading features file. Could not open file %1").arg(fileName);
		throw std::runtime_error(qPrintable(msg));
	}

	QTextStream in(&file);
	QString line = in.readLine();
	while (!line.isNull())
	{
		QStringList parts = line.split(' ');
		mapping[parts[0].toInt()] = coefficient * parts[1].toFloat();
		line = in.readLine();
	}
}

void
readMapInt(std::map<int, int>& mapping, QString folder, QString fileName)
{
	if (folder != "")
	{
		fileName = QDir(folder).filePath(fileName);
	}
	QFile file(fileName);
	if (!file.open(QIODevice::ReadOnly | QIODevice::Text))
	{
		const QString msg =
			QString("Error reading features file. Could not open file %1").arg(fileName);
		throw std::runtime_error(qPrintable(msg));
	}

	QTextStream in(&file);
	QString line = in.readLine();
	while (!line.isNull())
	{
		QStringList parts = line.split(' ');
		mapping[parts[0].toInt()] = parts[1].toInt();
		line = in.readLine();
	}
}


void
load(std::map<int, std::map<int, float> >& neuron_pre,
	float b1,
	std::map<int, std::map<int, float> >& neuron_postExc,
	float b2,
	std::map<int, std::map<int, float> >& neuron_postInh,
	std::map<int, float>& voxel_postAllExc,
	float b3,
	std::map<int, float>& voxel_postAllInh,
	std::map<int, int>& /*neuron_funct*/,
	std::map<int, std::set<int> >& /*voxel_neuronsPre*/,
	std::map<int, std::set<int> >& /*voxel_neuronsPostExc*/,
	std::map<int, std::set<int> >& /*voxel_neuronsPostInh*/)
{
	QDirIterator it_pre("features_pre");
	while (it_pre.hasNext())
	{
		QString file = it_pre.next();
		if (file.contains(".dat"))
		{
			QFileInfo fileInfo(file);
			int neuron = fileInfo.baseName().toInt();
			std::map<int, float> foo;
			readMapFloat(foo, "", file, b1);
			neuron_pre[neuron] = foo;
		}
	}
	//qDebug() << neuron_pre.size();

	QDirIterator it_postExc("features_postExc");
	while (it_postExc.hasNext())
	{
		QString file = it_postExc.next();
		if (file.contains(".dat"))
		{
			QFileInfo fileInfo(file);
			int neuron = fileInfo.baseName().toInt();
			std::map<int, float> foo;
			readMapFloat(foo, "", file, b2);
			neuron_postExc[neuron] = foo;
		}
	}

	QDirIterator it_postInh("features_postInh");
	while (it_postInh.hasNext())
	{
		QString file = it_postInh.next();
		if (file.contains(".dat"))
		{
			QFileInfo fileInfo(file);
			int neuron = fileInfo.baseName().toInt();
			std::map<int, float> foo;
			readMapFloat(foo, "", file,1);
			neuron_postInh[neuron] = foo;
		}
	}

	readMapFloat(voxel_postAllExc, "features_postAll", "voxel_postAllExc.dat", b3);
	readMapFloat(voxel_postAllInh, "features_postAll", "voxel_postAllInh.dat",1);
	/*
	readMapInt(neuron_funct, "features_meta", "neuron_funct.dat");

	readIndex(voxel_neuronsPre, "features_meta","voxel_neuronsPre.dat");
	readIndex(voxel_neuronsPostExc, "features_meta","voxel_neuronsPostExc.dat");
	readIndex(voxel_neuronsPostInh, "features_meta","voxel_neuronsPostInh.dat");
*/
}

/*
int main()
{
	fprintf(stderr, "addWithCuda failed!");
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
*/


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
